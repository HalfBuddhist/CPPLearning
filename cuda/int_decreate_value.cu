#include <stdio.h>

#include "hip/hip_runtime.h"


void addWithCuda(int *c);  // 1.定义传入的函数c

int main(void) {
  int c;
  c = 10;
  addWithCuda(&c);           // 2.传入参数变量（地址）
  hipDeviceReset();         // 6.重置CUDA设备释放程序占用的资源
  printf("Value is %d", c);  // 7.主机上打印显示数据

  system("pause");
  return 0;
}

__global__ void Decrease(int a, int b, int *c) { *c = a - b; }

void addWithCuda(int *c) {
  int *dev_c = 0;  //这个相当于内存和显存有一样的
  // 3.请求CUDA设备的内存（显存），执行CUDA函数
  hipMalloc((void **)&dev_c, sizeof(int));
  Decrease<<<1, 1>>>(15, 30, dev_c);
  // 4.等待设备所有线程任务执行完毕
  hipDeviceSynchronize();
  // 5.数据复制到主机，释放占用空间
  hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_c);
}
