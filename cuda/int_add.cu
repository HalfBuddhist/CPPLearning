// #include <Windows.h>
#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void Add(int i, int j) {
  int count;
  count = i + j;
  printf("\nNum is %d\n", count);
}

int main(void) {
  Add<<<1, 1>>>(10, 15);
  hipDeviceReset();  //重置CUDA设备释放程序占用的资源
  system("pause");
  return 0;
}