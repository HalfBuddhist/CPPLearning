#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void Decrease(int *a, int *b, int *c) { *c = *a - *b; }

void addWithCuda(int *c, int *a, int *b) {
  int *dev_c = 0;
  int *dev_a = 0;
  int *dev_b = 0;
  // 3.请求CUDA设备的内存（显存），执行CUDA函数
  hipMalloc((void **)&dev_c, sizeof(int));
  hipMalloc((void **)&dev_a, sizeof(int));
  hipMalloc((void **)&dev_b, sizeof(int));

  // 4.从主机复制数据到设备上
  hipMemcpy(dev_a, a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);

  Decrease<<<1, 1>>>(dev_a, dev_b, dev_c);

  // 5.等待设备所有线程任务执行完毕
  hipDeviceSynchronize();

  // 6.数据复制到主机，释放占用空间
  hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_c);
  hipFree(dev_a);
  hipFree(dev_b);
}

int main(void) {
  int c;
  int a, b;
  c = 10;
  a = 30;
  b = 15;
  addWithCuda(&c, &a, &b);   // 2.传入参数变量（地址）
  hipDeviceReset();         // 7.重置CUDA设备释放程序占用的资源
  printf("Value is %d", c);  // 8.主机上打印显示数据
  system("pause");
  return 0;
}